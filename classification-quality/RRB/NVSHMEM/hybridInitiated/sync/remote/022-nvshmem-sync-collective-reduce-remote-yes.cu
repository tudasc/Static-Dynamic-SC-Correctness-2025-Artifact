#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma write","rma write"],
    "RACE_PAIR": ["nvshmem_int_sum_reduce@33","nvshmem_int_put@83"],
    "CONSISTENCY_CALLS": ["nvshmem_quiet"],
    "SYNC_CALLS": ["nvshmem_barrier_all"],
    "NPROCS": 4,
    "DESCRIPTION": "Two conflicting operations nvshmem_int_sum_reduce and nvshmem_int_put without synchronization."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 4

__global__ void nvshmem_kernel(int *remote, int *localbuf, int *reduced) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;
    static uint64_t signal = 0;

    int my_pe = nvshmem_my_pe();   
    
    // CONFLICT
    nvshmem_int_sum_reduce(NVSHMEM_TEAM_WORLD, reduced, remote, 1);
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));
    int *reduced_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));
    hipMemset(reduced_d, 0, sizeof(int));

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Launch kernel normally 
    nvshmem_kernel<<<blocks, threads, shared_data_size>>>(remote_d, localbuf_d);

    if (my_pe == 0) {
        // CONFLICT
        nvshmem_int_put(remote_d, localbuf_d, 1, 1);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);
    nvshmem_free(reduced_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
