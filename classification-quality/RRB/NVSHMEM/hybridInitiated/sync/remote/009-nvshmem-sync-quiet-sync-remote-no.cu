#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "none",
    "ACCESS_SET": ["rma write","rma read"],
    "NPROCS": 2,
    "CONSISTENCY_CALLS": ["nvshmem_quiet"],
    "SYNC_CALLS": ["nvshmem_sync"],
    "DESCRIPTION": "Two conflicting operations put and get synchronized with nvshmem_quiet (consistency) and nvshmem_sync (process synchronization)."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    int my_pe = nvshmem_my_pe();
    if (my_pe == 0) {
        nvshmem_int_get(localbuf, remote, 1, 1);
    }
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

__global__ void nvshmem_sync() {
    nvshmem_quiet();
    nvshmem_sync(NVSHMEM_TEAM_WORLD);
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Launch kernel normally 
    nvshmem_kernel<<<blocks, threads, shared_data_size>>>(remote_d, localbuf_d);

    nvshmemx_collective_launch((void *)nvshmem_sync, blocks, threads, nullptr, shared_data_size, 0);

    if (my_pe == 0) {
        nvshmem_int_put(remote_d, myval_d, 1, 1);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
