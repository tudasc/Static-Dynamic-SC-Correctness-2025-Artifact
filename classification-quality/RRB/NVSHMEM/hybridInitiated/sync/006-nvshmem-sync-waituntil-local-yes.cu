#include "hip/hip_runtime.h"
/* Part of RMARaceBench, under BSD-3-Clause License
 * See https://github.com/RWTH-HPC/RMARaceBench/LICENSE for license information.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__global__ void nvshmem_kernel(int *remote, int *localbuf, uint64_t *flag) {
    // send data with signal (ping)
    // CONFLICT
    nvshmem_int_put_signal_nbi(remote, localbuf, 1, flag, 1, NVSHMEM_SIGNAL_SET, 1);
    // wait for pong from PE 1
    nvshmem_uint64_wait_until(flag, NVSHMEM_CMP_EQ, 1);
}

__global__ void nvshmem_kernel2(int *localbuf) {
    *localbuf = 1337;
}


__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

__global__ void nvshmem_quiet_kernelWrapper() {
    nvshmem_quiet();    
}


int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));
    uint64_t *flag_d = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d, flag_d};  // Kernel arguments
    void *args2[] = {localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));
    hipMemset(flag_d, 0, sizeof(int));

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    if (my_pe == 0) {
        hipMemset(localbuf_d, 42, sizeof(int));
        nvshmemx_collective_launch((const void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);
        // CONFLICT
        hipMemset(localbuf_d, 1337, sizeof(int));
    }

    if (my_pe == 1) {
        nvshmem_kernel2<<<blocks, threads, shared_data_size>>>(localbuf_d);
        // send data with signal (pong)
        nvshmem_int_put_signal(remote_d, localbuf_d, 1, flag_d, 1, NVSHMEM_SIGNAL_SET, 0);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);
    
    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
