#include "hip/hip_runtime.h"
/* Part of RMARaceBench, under BSD-3-Clause License
 * See https://github.com/RWTH-HPC/RMARaceBench/LICENSE for license information.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>
#include <string.h>

// Number of processing elements
#define PROC_NUM 2

__global__ void nvshmem_kernel(int *rem_ptr_alias, int *lbuf_ptr_alias) {
    int x = *lbuf_ptr_alias;

    nvshmem_barrier_all();
    *rem_ptr_alias += x;
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    int* rem_ptr_d = remote_d;
    int* lbuf_ptr_d = localbuf_d;
    int* rem_ptr_alias_d;
    int* lbuf_ptr_alias_d;

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    memcpy(&rem_ptr_alias_d, &rem_ptr_d, sizeof(int*));
    memcpy(&lbuf_ptr_alias_d, &lbuf_ptr_d, sizeof(int*));

    void *args[] = {rem_ptr_alias_d, lbuf_ptr_alias_d};  

    if (my_pe == 0) {
        nvshmem_int_put_nbi(rem_ptr_d, lbuf_ptr_d, 1, 1);
        nvshmemx_collective_launch((const void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
