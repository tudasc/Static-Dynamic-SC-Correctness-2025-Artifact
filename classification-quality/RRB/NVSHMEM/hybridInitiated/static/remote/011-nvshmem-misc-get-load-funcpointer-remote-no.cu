#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "none",
    "ACCESS_SET": ["rma read","load"],
    "NPROCS": 2,
    "DESCRIPTION": "Two non-conflicting operations get and load executed concurrently with no race."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__device__  __attribute__((noinline)) void rank0(int* rem_ptr, int* lbuf_ptr, int* x) {
    nvshmem_int_get(lbuf_ptr, rem_ptr, 1, 1);
}

__attribute__((noinline)) void rank1(int* rem_ptr, int* lbuf_ptr, int* x) {
    hipMemcpy(x, rem_ptr, sizeof(int), hipMemcpyDeviceToHost);
    printf("*remote is %d\n", *x);
}

__attribute__((noinline)) void rank0_help(int* rem_ptr, int* lbuf_ptr, int* x, int* shared_data_size) {
    dim3 blocks(1);
    dim3 threads(1);
    void *args[] = {rem_ptr, lbuf_ptr, x};

    nvshmemx_collective_launch((void *)rank0, blocks, threads, args, shared_data_size, 0);
}

__attribute__((noinline)) void rank1_help(int* rem_ptr, int* lbuf_ptr, int* x, int* shared_data_size) {
    rank1(rem_ptr, lbuf_ptr, x);
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    int* rem_ptr_d = remote_d;
    int* lbuf_ptr_d = localbuf_d;

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    void (*rankfunc)(int* rem_ptr, int* lbuf_ptr, int* x, int* shared_data_size);

    if (my_pe == 0) {
        rankfunc = rank0_help;
    }

    if (my_pe == 1) {
        rankfunc = rank1_help;
    }

    (*rankfunc)(rem_ptr_d, lbuf_ptr_d, &remote, &shared_data_size);

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
