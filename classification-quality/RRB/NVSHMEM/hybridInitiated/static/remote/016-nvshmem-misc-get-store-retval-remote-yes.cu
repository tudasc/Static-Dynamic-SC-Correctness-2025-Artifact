#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma read","store"],
    "RACE_PAIR": ["shmem_int_get@22","hipMemcpy@85"],
    "NPROCS": 2,
    "DESCRIPTION": "Two conflicting operations get and store executed concurrently which leads to a race."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__device__  __attribute__((noinline)) void rank0(int* rem_ptr, int* lbuf_ptr, int* x) {
    nvshmem_int_get(lbuf_ptr, rem_ptr, 1, 1);
}

__attribute__((noinline)) int* aliasgenerator(int** x) { return *x; }

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    int* rem_ptr = &remote;
    int* rem_ptr_d = remote_d;
    int* lbuf_ptr_d = localbuf_d;
    int* rem_ptr_alias;
    int* rem_ptr_alias_d;
    int* lbuf_ptr_alias_d;
    
    rem_ptr_alias = aliasgenerator(&rem_ptr);
    rem_ptr_alias_d = aliasgenerator(&rem_ptr_d);
    lbuf_ptr_alias_d = aliasgenerator(&lbuf_ptr_d);

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    if (my_pe == 0) {
        // CONFLICT
        nvshmemx_collective_launch((void *)rank0, blocks, threads, args, shared_data_size, 0);
    }

    if (my_pe == 1) {
        // CONFLICT
        hipMemcpy(rem_ptr_alias_d, rem_ptr_alias, sizeof(int), hipMemcpyHostToDevice);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
