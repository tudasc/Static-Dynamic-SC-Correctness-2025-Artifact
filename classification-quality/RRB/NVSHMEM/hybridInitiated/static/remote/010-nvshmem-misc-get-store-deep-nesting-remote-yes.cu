#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma read","store"],
    "RACE_PAIR": ["nvshmem_int_get@23","STORE@90"],
    "NPROCS": 2,
    "DESCRIPTION": "Two conflicting operations get and store executed concurrently which leads to a race."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__device__  __attribute__((noinline)) void deeeeeeeeep(int* rem_ptr, int* lbuf_ptr) {
    // CONFLICT
    nvshmem_int_get(lbuf_ptr, rem_ptr, 1, 1);
}

__device__  __attribute__((noinline)) void deeeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeep(int* rem_ptr, int* lbuf_ptr) { deeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeep(int* rem_ptr, int* lbuf_ptr) { deeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deep(int* rem_ptr, int* lbuf_ptr) { deeep(rem_ptr, lbuf_ptr); }

__global__ void rank0(int* rem_ptr, int* lbuf_ptr)
{
    deep(rem_ptr, lbuf_ptr);
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    int* rem_ptr_d = remote_d;
    int* lbuf_ptr_d = localbuf_d;

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    void *args[] = {rem_ptr_alias_d, lbuf_ptr_alias_d};

    if (my_pe == 0) {
        nvshmemx_collective_launch((void *)rank0, blocks, threads, args, shared_data_size, 0);
    }

    if (my_pe == 1) {
        // CONFLICT
        hipMemcpy(remote_d, remote, sizeof(int), hipMemcpyHostToDevice);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
