#include "hip/hip_runtime.h"
/* Part of RMARaceBench, under BSD-3-Clause License
 * See https://github.com/RWTH-HPC/RMARaceBench/LICENSE for license information.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__device__  __attribute__((noinline)) void deeeeeeeeep(int* rem_ptr, int* lbuf_ptr) { nvshmem_int_put_nbi(rem_ptr, lbuf_ptr, 1, 1); }

__device__  __attribute__((noinline)) void deeeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeep(int* rem_ptr, int* lbuf_ptr) { deeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeep(int* rem_ptr, int* lbuf_ptr) { deeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deep(int* rem_ptr, int* lbuf_ptr) { deeep(rem_ptr, lbuf_ptr); }

__global__ void rank0(int* rem_ptr, int* lbuf_ptr)
{
    deep(rem_ptr, lbuf_ptr);
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    int* rem_ptr_d = remote_d;
    int* lbuf_ptr_d = localbuf_d;

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    if (my_pe == 0) {
        // Launch kernel normally 
        rank0<<<blocks, threads, shared_data_size>>>(rem_ptr_d, lbuf_ptr_d);
        // Non-Conflicting load
        hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);
        printf("localbuf is %d\n", localbuf);
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
