#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma read","load"],
    "RACE_PAIR": ["nvshmem_int_g@25","hipMemcpy@74"],
    "NPROCS": 2,
    "DESCRIPTION": "Two conflicting operations g and load executed concurrently which leads to a race."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    int my_pe = nvshmem_my_pe();
    if (my_pe == 0) {
        // CONFLICT
        nvshmem_int_p(remote, 42, 1);
    }
}

__global__ void nvshmem_barrier_all_kernelWrapper() {
    nvshmem_barrier_all();    
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    dim3 blocks(1);
    dim3 threads(1);

    // Initialize memory
    hipMemset(remote_d, 0, sizeof(int));
    hipMemset(localbuf_d, 1, sizeof(int));

    // Synchronize across all PEs
    nvshmem_barrier_all();    
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Launch kernel normally 
    nvshmem_kernel<<<blocks, threads, shared_data_size>>>(remote_d, localbuf_d);

    if (my_pe == 1) {
        // CONFLICT
        hipMemcpy(remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
        printf("remotebuf is %d\n", remote);
    }

    // Synchronize
    nvshmem_barrier_all();
    nvshmemx_collective_launch((const void *)nvshmem_barrier_all_kernelWrapper, blocks, threads, nullptr, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
