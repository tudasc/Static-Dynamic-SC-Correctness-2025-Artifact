#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma read","store"],
    "RACE_PAIR": ["nvshmem_int_get@23","STORE@59"],
    "NPROCS": 2,
    "DESCRIPTION": "Two conflicting operations get and store executed concurrently which leads to a race."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__device__  __attribute__((noinline)) void deeeeeeeeep(int* rem_ptr, int* lbuf_ptr) {
    // CONFLICT
    nvshmem_int_get(lbuf_ptr, rem_ptr, 1, 1); 
}

__device__  __attribute__((noinline)) void deeeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeep(int* rem_ptr, int* lbuf_ptr) { deeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeep(int* rem_ptr, int* lbuf_ptr) { deeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deep(int* rem_ptr, int* lbuf_ptr) { deeep(rem_ptr, lbuf_ptr); }

__device__ void rank0(int* rem_ptr, int* lbuf_ptr)
{
    deep(rem_ptr, lbuf_ptr);
}

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;

    int* rem_ptr = remote;
    int* lbuf_ptr = localbuf;

    int my_pe = nvshmem_my_pe();

    // Synchronize across all PEs
    nvshmem_barrier_all();    

    if (my_pe == 0) {
        rank0(rem_ptr, lbuf_ptr);
    }

    if (my_pe == 1) {
        // CONFLICT
        *lbuf_ptr = 42;
    }

    nvshmem_barrier_all();
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Step 3: Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Step 4: Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Launch kernel collectively across all PEs
    nvshmemx_collective_launch((void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize
    nvshmem_barrier_all();

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}