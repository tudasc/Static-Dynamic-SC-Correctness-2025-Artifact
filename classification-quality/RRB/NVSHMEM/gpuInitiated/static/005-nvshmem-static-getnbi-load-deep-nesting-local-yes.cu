#include "hip/hip_runtime.h"
/* Part of RMARaceBench, under BSD-3-Clause License
 * See https://github.com/RWTH-HPC/RMARaceBench/LICENSE for license information.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__device__  __attribute__((noinline)) void deeeeeeeeep(int* rem_ptr, int* lbuf_ptr) {
    /* conflicting getnbi and load */
    // CONFLICT
    nvshmem_int_get_nbi(lbuf_ptr, rem_ptr, 1, 1); 
}

__device__  __attribute__((noinline)) void deeeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeeep(int* rem_ptr, int* lbuf_ptr) { deeeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeeep(int* rem_ptr, int* lbuf_ptr) { deeeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deeep(int* rem_ptr, int* lbuf_ptr) { deeeep(rem_ptr, lbuf_ptr); }
__device__  __attribute__((noinline)) void deep(int* rem_ptr, int* lbuf_ptr) { deeep(rem_ptr, lbuf_ptr); }

__device__ void rank0(int* rem_ptr, int* lbuf_ptr, int *x)
{
    deep(rem_ptr, lbuf_ptr);
    // CONFLICT
    *x = *lbuf_ptr;
}

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;

    int* rem_ptr = remote;
    int* lbuf_ptr = localbuf;

    int my_pe = nvshmem_my_pe();

    // Synchronize across all PEs
    nvshmem_barrier_all();    

    if (my_pe == 0) {
        rank0(rem_ptr, lbuf_ptr, &x);
    } 

    nvshmem_barrier_all();
    *remote += x;
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Step 3: Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Step 4: Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Launch kernel collectively across all PEs
    nvshmemx_collective_launch((const void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}