#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma atomic write","load"],
    "RACE_PAIR": ["nvshmem_int_atomic_set@35","LOAD@40"],
    "NPROCS": 2,
    "DESCRIPTION": "Two conflicting operations atomicset and load executed concurrently which leads to a race."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mynvshmem.h>
#include <mynvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;

    int my_pe = nvshmem_my_pe();

    // Synchronize across all PEs
    nvshmem_barrier_all();

    if (my_pe == 0) {
        // CONFLICT
        nvshmem_int_atomic_set(remote, 42, 1);
    }

    if (my_pe == 1) {
        // CONFLICT
        x = *remote;
    }

    // Synchronize across all PEs
    nvshmem_barrier_all();
    *remote += x;
}

int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Launch kernel collectively across all PEs
    nvshmemx_collective_launch((const void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize
    nvshmem_barrier_all();

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}