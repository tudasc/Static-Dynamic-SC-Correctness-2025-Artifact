#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "none",
    "ACCESS_SET": ["rma write","rma write"],
    "CONSISTENCY_CALLS": ["nvshmem_quiet"],
    "SYNC_CALLS": ["nvshmem_barrier_all"],
    "NPROCS": 4,
    "DESCRIPTION": "Two conflicting operations nvshmem_int_sum_reduce and nvshmem_int_put synchronized through nvshmem_barrier_all."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 4

__global__ void nvshmem_kernel(int *remote, int *localbuf, int *reduced) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;
    static uint64_t signal = 0;

    int my_pe = nvshmem_my_pe();

    // Synchronize across all PEs
    nvshmem_barrier_all();   
    
    // Potential conflict
    nvshmem_int_sum_reduce(NVSHMEM_TEAM_WORLD, reduced, remote, 1);

    nvshmem_barrier_all();

    if (my_pe == 0) {
        // Potential conflict
        nvshmem_int_put(remote, localbuf, 1, 1);
    }

    nvshmem_barrier_all();
}


int main(int argc, char **argv) {
    int remote, localbuf, reduced;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));
    int *reduced_d = (int *)nvshmem_malloc(sizeof(int));

    // Step 3: Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Step 4: Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Launch kernel collectively across all PEs
    nvshmemx_collective_launch((void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&reduced, reduced_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize
    nvshmem_barrier_all();

    printf("PE %d: localbuf = %d, remote = %d, reduced = %d\n", my_pe, localbuf, remote, reduced);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);
    nvshmem_free(reduced_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
