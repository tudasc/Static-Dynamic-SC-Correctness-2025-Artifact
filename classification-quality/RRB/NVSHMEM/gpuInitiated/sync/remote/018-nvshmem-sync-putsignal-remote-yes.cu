#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma write","local read"],
    "RACE_PAIR": ["LOAD@36","nvshmem_int_put_signal@47"],
    "CONSISTENCY_CALLS": ["nvshmem_barrier_all"],
    "SYNC_CALLS": ["nvshmem_signal_fetch"],
    "NPROCS": 2,
    "DESCRIPTION": "Signalled put where PE 0 polls with plain LOADs on the signal instead of using nvshmem_signal_fetch."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 2

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;
    static uint64_t signal = 0;

    int my_pe = nvshmem_my_pe();

    // Synchronize across all PEs
    nvshmem_barrier_all();    

    if (my_pe == 0) {
        // CONFLICT
        while (signal < PROC_NUM - 1) {
            continue;
        }

        x = *remote;
    }

    if (my_pe == 1) {
        *localbuf = my_pe;
        // CONFLICT
        nvshmem_int_put_signal(remote, localbuf, 1, &signal, 1, NVSHMEM_SIGNAL_ADD, 0);
    }

    nvshmem_barrier_all();
}


int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Step 3: Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Step 4: Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Launch kernel collectively across all PEs
    nvshmemx_collective_launch((void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize
    nvshmem_barrier_all();

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
