#include "hip/hip_runtime.h"
// RACE LABELS BEGIN
/*
{
    "RACE_KIND": "remote",
    "ACCESS_SET": ["rma write","load"],
    "RACE_PAIR": ["nvshmem_int_put@43","LOAD@53"],
    "CONSISTENCY_CALLS": ["nvshmem_quiet"],
    "SYNC_CALLS": ["nvshmem_team_sync"],
    "NPROCS": 4,
    "DESCRIPTION": "PE 2 part of the team puts to PE 3 which is *not* part of the team. There is no synchronization between PE 2 and PE3, since they are not in the same team."
}
*/
// RACE LABELS END

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>

// Number of processing elements
#define PROC_NUM 4

__global__ void nvshmem_kernel(int *remote, int *localbuf) {
    // Initialize memory
    *remote = 0;
    *localbuf = 1;
    int x = 0;

    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();

    // Synchronize across all PEs
    nvshmem_barrier_all();    

    nvshmem_team_t team = NVSHMEM_TEAM_INVALID;
    nvshmem_team_split_strided(NVSHMEM_TEAM_WORLD, 0, 2, num_pe / 2, NULL, 0lu, &team);

    nvshmem_barrier_all();

    if (NVSHMEM_TEAM_INVALID != team) {
        if (my_pe == 2) {
            // CONFLICT
            nvshmem_int_put(remote, localbuf, 1, 3); // P2 puts to P3
        }
        nvshmem_quiet(); // Synchronisation

        nvshmem_team_sync(team); // Synchronisation
    }

    if (NVSHMEM_TEAM_INVALID == team) {
        if (my_pe == 3)
        // CONFLICT
        x = *remote;
    }

    nvshmem_barrier_all();
}


int main(int argc, char **argv) {
    int remote, localbuf;

    // Initialize NVSHMEM
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);

    // Get the number of PEs and the current PE's rank
    int my_pe = nvshmem_my_pe();
    int num_pe = nvshmem_n_pes();
    // Ensure the required number of PEs
    if (num_pe != PROC_NUM) {
        printf("Got %d PEs, expected %d\n", num_pe, PROC_NUM);
        nvshmem_global_exit(1);
    }

    // Allocate symmetric memory on the device
    int *remote_d = (int *)nvshmem_malloc(sizeof(int));
    int *localbuf_d = (int *)nvshmem_malloc(sizeof(int));

    // Step 3: Allocate shared memory across PEs
    size_t shared_data_size = 0 * sizeof(int);

    // Step 4: Define kernel execution parameters
    void *args[] = {remote_d, localbuf_d};  // Kernel arguments
    dim3 blocks(1);
    dim3 threads(1);

    // Launch kernel collectively across all PEs
    nvshmemx_collective_launch((void *)nvshmem_kernel, blocks, threads, args, shared_data_size, 0);

    // Copy data back to host
    hipMemcpy(&remote, remote_d, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&localbuf, localbuf_d, sizeof(int), hipMemcpyDeviceToHost);

    // Synchronize
    nvshmem_barrier_all();

    printf("PE %d: localbuf = %d, remote = %d\n", my_pe, localbuf, remote);

    // Synchronize again
    nvshmem_barrier_all();

    printf("Process %d: Execution finished, variable contents: remote = %d, localbuf = %d\n", my_pe, remote, localbuf);

    // Free NVSHMEM symmetric memory
    nvshmem_free(remote_d);
    nvshmem_free(localbuf_d);

    // Finalize NVSHMEM
    nvshmem_finalize();

    return 0;
}
