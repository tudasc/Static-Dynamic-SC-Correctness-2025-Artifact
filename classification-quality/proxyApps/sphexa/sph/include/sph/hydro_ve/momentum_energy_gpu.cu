#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cub.hpp"
#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/primitives/warpscan.cuh"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_ve/momentum_energy_kern.hpp"

namespace sph
{
namespace cuda
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

static __device__ float minDt_ve_device;

template<bool avClean, class Tc, class Tm, class T, class Tm1, class KeyType>
__global__ void momentumEnergyGpu(Tc K, Tc Kcour, T Atmin, T Atmax, T ramp, unsigned ngmax, const cstone::Box<Tc> box,
                                  const LocalIndex* grpStart, const LocalIndex* grpEnd, LocalIndex numGroups,
                                  const cstone::OctreeNsView<Tc, KeyType> tree, const Tc* x, const Tc* y, const Tc* z,
                                  const T* vx, const T* vy, const T* vz, const T* h, const Tm* m, const T* prho,
                                  const T* tdpdTrho, const T* c, const T* c11, const T* c12, const T* c13, const T* c22,
                                  const T* c23, const T* c33, const T* wh, const T* kx, const T* xm, const T* alpha,
                                  const T* dV11, const T* dV12, const T* dV13, const T* dV22, const T* dV23,
                                  const T* dV33, T* grad_P_x, T* grad_P_y, T* grad_P_z, Tm1* du, LocalIndex* nidx,
                                  TreeNodeIndex* globalPool, float* groupDt)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    T dt_i = INFINITY;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) { break; }

        LocalIndex bodyBegin = grpStart[targetIdx];
        LocalIndex bodyEnd   = grpEnd[targetIdx];
        LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);
        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        T        maxvsignal;

        if (i < bodyEnd)
        {
            momentumAndEnergyJLoop<avClean, TravConfig::targetSize>(
                i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, vx, vy, vz, h, m, prho, tdpdTrho, c, c11, c12,
                c13, c22, c23, c33, Atmin, Atmax, ramp, wh, kx, xm, alpha, dV11, dV12, dV13, dV22, dV23, dV33, grad_P_x,
                grad_P_y, grad_P_z, du, &maxvsignal);
        }

        auto dt_lane = (i < bodyEnd) ? tsKCourant(maxvsignal, h[i], c[i], Kcour) : INFINITY;
        if (groupDt != nullptr)
        {
            auto min_dt_group = cstone::warpMin(dt_lane);
            if ((threadIdx.x & (GpuConfig::warpSize - 1)) == 0)
            {
                groupDt[targetIdx] = stl::min(groupDt[targetIdx], min_dt_group);
            }
        }

        dt_i = stl::min(dt_i, dt_lane);
    }

    typedef hipcub::BlockReduce<T, TravConfig::numThreads> BlockReduce;
    __shared__ typename BlockReduce::TempStorage        temp_storage;

    BlockReduce reduce(temp_storage);
    T           blockMin = reduce.Reduce(dt_i, hipcub::Min());
    __syncthreads();

    if (threadIdx.x == 0) { cstone::atomicMinFloat(&minDt_ve_device, blockMin); }
}

template<bool avClean, class Dataset>
void computeMomentumEnergy(const GroupView& grp, float* groupDt, Dataset& d,
                           const cstone::Box<typename Dataset::RealType>& box)
{
    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, d.ngmax);

    float huge = 1e10;
    checkGpuErrors(hipMemcpyToSymbol(GPU_SYMBOL(minDt_ve_device), &huge, sizeof(huge)));
    cstone::resetTraversalCounters<<<1, 1>>>();

    momentumEnergyGpu<avClean><<<TravConfig::numBlocks(), TravConfig::numThreads>>>(
        d.K, d.Kcour, d.Atmin, d.Atmax, d.ramp, d.ngmax, box, grp.groupStart, grp.groupEnd, grp.numGroups, d.treeView,
        rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.vx), rawPtr(d.devData.vy),
        rawPtr(d.devData.vz), rawPtr(d.devData.h), rawPtr(d.devData.m), rawPtr(d.devData.prho),
        rawPtr(d.devData.tdpdTrho), rawPtr(d.devData.c), rawPtr(d.devData.c11), rawPtr(d.devData.c12),
        rawPtr(d.devData.c13), rawPtr(d.devData.c22), rawPtr(d.devData.c23), rawPtr(d.devData.c33),
        rawPtr(d.devData.wh), rawPtr(d.devData.kx), rawPtr(d.devData.xm), rawPtr(d.devData.alpha),
        rawPtr(d.devData.dV11), rawPtr(d.devData.dV12), rawPtr(d.devData.dV13), rawPtr(d.devData.dV22),
        rawPtr(d.devData.dV23), rawPtr(d.devData.dV33), rawPtr(d.devData.ax), rawPtr(d.devData.ay),
        rawPtr(d.devData.az), rawPtr(d.devData.du), nidxPool, traversalPool, groupDt);
    checkGpuErrors(hipGetLastError());

    float minDt;
    checkGpuErrors(hipMemcpyFromSymbol(&minDt, HIP_SYMBOL(GPU_SYMBOL(minDt_ve_device)), sizeof(minDt)));
    d.minDtCourant = minDt;
}

#define MOM_ENERGY(avc)                                                                                                \
    template void computeMomentumEnergy<avc>(const GroupView& grp, float*, sphexa::ParticlesData<cstone::GpuTag>& d,   \
                                             const cstone::Box<SphTypes::CoordinateType>&)

MOM_ENERGY(true);
MOM_ENERGY(false);

} // namespace cuda
} // namespace sph
