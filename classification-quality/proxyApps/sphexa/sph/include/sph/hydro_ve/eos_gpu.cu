#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/primitives/math.hpp"
#include "cstone/util/tuple.hpp"

#include "sph/sph_gpu.hpp"
#include "sph/eos.hpp"
#include "sph/particles_data.hpp"

namespace sph
{
namespace cuda
{

template<class Tt, class Tm, class Thydro>
__global__ void cudaEOS(size_t firstParticle, size_t lastParticle, Tm mui, Tt gamma, const Tt* temp, const Tm* m,
                        const Thydro* kx, const Thydro* xm, const Thydro* gradh, Thydro* prho, Thydro* c, Thydro* rho,
                        Thydro* p)
{
    unsigned i = firstParticle + blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lastParticle) return;

    Thydro p_i;
    Thydro rho_i         = kx[i] * m[i] / xm[i];
    util::tie(p_i, c[i]) = idealGasEOS(temp[i], rho_i, mui, gamma);
    prho[i]              = p_i / (kx[i] * m[i] * m[i] * gradh[i]);
    if (rho) { rho[i] = rho_i; }
    if (p) { p[i] = p_i; }
}

template<class Tt, class Tm, class Thydro>
void computeEOS(size_t firstParticle, size_t lastParticle, Tm mui, Tt gamma, const Tt* temp, const Tm* m,
                const Thydro* kx, const Thydro* xm, const Thydro* gradh, Thydro* prho, Thydro* c, Thydro* rho,
                Thydro* p)
{
    if (firstParticle == lastParticle) { return; }
    unsigned numThreads = 256;
    unsigned numBlocks  = cstone::iceil(lastParticle - firstParticle, numThreads);
    cudaEOS<<<numBlocks, numThreads>>>(firstParticle, lastParticle, mui, gamma, temp, m, kx, xm, gradh, prho, c, rho,
                                       p);
    checkGpuErrors(hipDeviceSynchronize());
}

#define COMPUTE_EOS(Ttemp, Tm, Thydro)                                                                                 \
    template void computeEOS(size_t firstParticle, size_t lastParticle, Tm mui, Ttemp gamma, const Ttemp* temp,        \
                             const Tm* m, const Thydro* kx, const Thydro* xm, const Thydro* gradh, Thydro* prho,       \
                             Thydro* c, Thydro* rho, Thydro* p)

COMPUTE_EOS(double, double, double);
COMPUTE_EOS(double, float, double);
COMPUTE_EOS(double, float, float);
COMPUTE_EOS(float, float, float);

template<typename Th, typename Tu>
__global__ void cudaComputeIsothermalEOS(size_t first, size_t last, Th cConst, Th* rho, Th* p, const Th* m,
                                         const Th* kx, const Th* xm, const Th* gradh, Th* prho, Tu* temp)
{
    unsigned i = first + blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= last) return;

    Th rho_i = kx[i] * m[i] / xm[i];
    Th p_i   = isothermalEOS(cConst, rho_i);
    prho[i]  = p_i / (kx[i] * m[i] * m[i] * gradh[i]);
    if (rho) { rho[i] = rho_i; }
    if (p) { p[i] = p_i; }
    if (temp) { temp[i] = 0; }
}

template<typename Dataset>
void computeIsothermalEOS(size_t first, size_t last, Dataset& d)
{
    if (first == last) { return; }
    unsigned numThreads = 256;
    unsigned numBlocks  = cstone::iceil(last - first, numThreads);
    cudaComputeIsothermalEOS<<<numBlocks, numThreads>>>(first, last, d.soundSpeedConst, rawPtr(d.devData.rho),
                                                        rawPtr(d.devData.p), rawPtr(d.devData.m), rawPtr(d.devData.kx),
                                                        rawPtr(d.devData.xm), rawPtr(d.devData.gradh),
                                                        rawPtr(d.devData.prho), rawPtr(d.devData.temp));
    checkGpuErrors(hipDeviceSynchronize());
}

template void computeIsothermalEOS(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>& d);

} // namespace cuda
} // namespace sph
