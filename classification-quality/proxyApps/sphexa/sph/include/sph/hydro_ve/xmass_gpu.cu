#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2024 CSCS, ETH Zurich, University of Basel, University of Zurich
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <thrust/transform.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_ve/xmass_kern.hpp"

namespace sph
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::NcStats;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

unsigned nsGroupSize() { return TravConfig::targetSize; }

namespace cuda
{

__device__ bool nc_h_convergenceFailure = false;

template<class Tc, class Tm, class T, class KeyType>
__global__ void xmassGpu(Tc K, unsigned ng0, unsigned ngmax, const cstone::Box<Tc> box, const LocalIndex* grpStart,
                         const LocalIndex* grpEnd, LocalIndex numGroups, const cstone::OctreeNsView<Tc, KeyType> tree,
                         unsigned* nc, const Tc* x, const Tc* y, const Tc* z, T* h, const Tm* m, const T* wh,
                         const T* whd, T* xm, LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) return;

        LocalIndex bodyBegin = grpStart[targetIdx];
        LocalIndex bodyEnd   = grpEnd[targetIdx];
        LocalIndex i         = bodyBegin + laneIdx;

        unsigned ncSph =
            1 + traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool)[0];

        constexpr int ncMaxIteration = 9;
        for (int ncIt = 0; ncIt <= ncMaxIteration; ++ncIt)
        {
            bool repeat = (ncSph < ng0 / 4 || (ncSph - 1) > ngmax) && i < bodyEnd;
            if (!cstone::ballotSync(repeat)) { break; }
            if (repeat) { h[i] = updateH(ng0, ncSph, h[i]); }
            ncSph =
                1 + traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool)[0];

            if (ncIt == ncMaxIteration) { nc_h_convergenceFailure = true; }
        }

        if (i >= bodyEnd) continue;

        unsigned ncCapped = stl::min(ncSph - 1, ngmax);
        xm[i] = sph::xmassJLoop<TravConfig::targetSize>(i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, h, m, wh,
                                                        whd);
        nc[i] = ncSph;
    }
}

template<class Dataset>
void computeXMass(const GroupView& grp, Dataset& d, const cstone::Box<typename Dataset::RealType>& box)
{
    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, d.ngmax);
    cstone::resetTraversalCounters<<<1, 1>>>();

    xmassGpu<<<TravConfig::numBlocks(), TravConfig::numThreads>>>(
        d.K, d.ng0, d.ngmax, box, grp.groupStart, grp.groupEnd, grp.numGroups, d.treeView, rawPtr(d.devData.nc),
        rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.h), rawPtr(d.devData.m),
        rawPtr(d.devData.wh), rawPtr(d.devData.whd), rawPtr(d.devData.xm), nidxPool, traversalPool);
    checkGpuErrors(hipDeviceSynchronize());

    NcStats::type stats[NcStats::numStats];
    checkGpuErrors(hipMemcpyFromSymbol(stats, HIP_SYMBOL(GPU_SYMBOL(cstone::ncStats)), NcStats::numStats * sizeof(NcStats::type)));

    bool convergenceFailure;
    checkGpuErrors(hipMemcpyFromSymbol(&convergenceFailure, HIP_SYMBOL(GPU_SYMBOL(nc_h_convergenceFailure)), sizeof(bool)));

    NcStats::type maxP2P   = stats[cstone::NcStats::maxP2P];
    NcStats::type maxStack = stats[cstone::NcStats::maxStack];

    d.devData.stackUsedNc = maxStack;

    if (maxP2P == 0xFFFFFFFF) { throw std::runtime_error("GPU traversal stack exhausted in neighbor search\n"); }
    if (convergenceFailure) { throw std::runtime_error("coupled nc/h-updated failed to converge"); }
}

template void computeXMass(const GroupView& grp, sphexa::ParticlesData<cstone::GpuTag>& d,
                           const cstone::Box<SphTypes::CoordinateType>&);

template<class Tm, class Trho>
__global__ void convertXmassToRho(const LocalIndex* grpStart, const LocalIndex* grpEnd, LocalIndex numGroups,
                                  const Tm* m, Trho* rho)
{
    LocalIndex tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= numGroups) { return; }

    LocalIndex bodyBegin = grpStart[tid];
    LocalIndex bodyEnd   = grpEnd[tid];

    for (auto i = bodyBegin; i < bodyEnd; ++i)
    {
        rho[i] = m[i] / rho[i];
    }
}

template<class Dataset>
void computeDensity(const GroupView& grp, Dataset& d, const cstone::Box<typename Dataset::RealType>& box)
{
    swap(d.devData.xm, d.devData.rho);
    computeXMass(grp, d, box);
    swap(d.devData.xm, d.devData.rho);

    unsigned numThreads = 256;
    unsigned numBlocks  = (grp.numGroups + numThreads - 1) / numThreads;
    if (numBlocks == 0) { return; }

    // rho[i] = m[i] / rho[i];
    convertXmassToRho<<<numBlocks, numThreads>>>(grp.groupStart, grp.groupEnd, grp.numGroups, rawPtr(d.devData.m),
                                                 rawPtr(d.devData.rho));
}

template void computeDensity(const GroupView&, sphexa::ParticlesData<cstone::GpuTag>& d,
                             const cstone::Box<SphTypes::CoordinateType>&);

} // namespace cuda
} // namespace sph
