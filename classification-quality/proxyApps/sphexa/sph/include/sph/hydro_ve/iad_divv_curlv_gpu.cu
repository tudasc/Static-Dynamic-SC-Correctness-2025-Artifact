#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_ve/iad_kern.hpp"
#include "sph/hydro_ve/divv_curlv_kern.hpp"

namespace sph
{
namespace cuda
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

template<class Tc, class T, class KeyType>
__global__ void
iadDivvCurlvGpu(Tc K, unsigned ngmax, const cstone::Box<Tc> box, const LocalIndex* grpStart, const LocalIndex* grpEnd,
                LocalIndex numGroups, const cstone::OctreeNsView<Tc, KeyType> tree, const Tc* x, const Tc* y,
                const Tc* z, const T* vx, const T* vy, const T* vz, const T* h, const T* wh, const T* whd, const T* xm,
                const T* kx, T* c11, T* c12, T* c13, T* c22, T* c23, T* c33, T* divv, T* curlv, T* dV11, T* dV12,
                T* dV13, T* dV22, T* dV23, T* dV33, cstone::LocalIndex* nidx, TreeNodeIndex* globalPool, bool doGradV)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    cstone::LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) return;

        LocalIndex bodyBegin = grpStart[targetIdx];
        LocalIndex bodyEnd   = grpEnd[targetIdx];
        LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        if (i >= bodyEnd) continue;

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        IADJLoop<TravConfig::targetSize>(i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, h, wh, whd, xm, kx, c11,
                                         c12, c13, c22, c23, c33);
        divV_curlVJLoop<TravConfig::targetSize>(i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, vx, vy, vz, h,
                                                c11, c12, c13, c22, c23, c33, wh, whd, kx, xm, divv, curlv, dV11, dV12,
                                                dV13, dV22, dV23, dV33, doGradV);
    }
}

template<class Dataset>
void computeIadDivvCurlv(const GroupView& grp, Dataset& d, const cstone::Box<typename Dataset::RealType>& box)
{
    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, d.ngmax);
    cstone::resetTraversalCounters<<<1, 1>>>();

    bool  doGradV = d.devData.x.size() == d.devData.dV11.size();
    auto* d_curlv = (d.devData.x.size() == d.devData.curlv.size()) ? rawPtr(d.devData.curlv) : nullptr;

    iadDivvCurlvGpu<<<TravConfig::numBlocks(), TravConfig::numThreads>>>(
        d.K, d.ngmax, box, grp.groupStart, grp.groupEnd, grp.numGroups, d.treeView, rawPtr(d.devData.x),
        rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.vx), rawPtr(d.devData.vy), rawPtr(d.devData.vz),
        rawPtr(d.devData.h), rawPtr(d.devData.wh), rawPtr(d.devData.whd), rawPtr(d.devData.xm), rawPtr(d.devData.kx),
        rawPtr(d.devData.c11), rawPtr(d.devData.c12), rawPtr(d.devData.c13), rawPtr(d.devData.c22),
        rawPtr(d.devData.c23), rawPtr(d.devData.c33), rawPtr(d.devData.divv), d_curlv, rawPtr(d.devData.dV11),
        rawPtr(d.devData.dV12), rawPtr(d.devData.dV13), rawPtr(d.devData.dV22), rawPtr(d.devData.dV23),
        rawPtr(d.devData.dV33), nidxPool, traversalPool, doGradV);
    checkGpuErrors(hipDeviceSynchronize());
}

template void computeIadDivvCurlv(const GroupView& grp, sphexa::ParticlesData<cstone::GpuTag>& d,
                                  const cstone::Box<SphTypes::CoordinateType>&);

} // namespace cuda
} // namespace sph
