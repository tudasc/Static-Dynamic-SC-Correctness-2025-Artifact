#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Pressure gradient (momentum) and energy i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cub.hpp"
#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/primitives/warpscan.cuh"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_std/momentum_energy_kern.hpp"

namespace sph
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

static __device__ float minDt_device;

template<class Tc, class Tm, class T, class Tm1, class KeyType>
__global__ void cudaGradP(Tc K, Tc Kcour, unsigned ngmax, cstone::Box<Tc> box, const LocalIndex* grpStart,
                          const LocalIndex* grpEnd, LocalIndex numGroups, const cstone::OctreeNsView<Tc, KeyType> tree,
                          const Tc* x, const Tc* y, const Tc* z, const T* vx, const T* vy, const T* vz, const T* h,
                          const Tm* m, const T* rho, const T* p, const T* c, const T* c11, const T* c12, const T* c13,
                          const T* c22, const T* c23, const T* c33, const T* wh, const T* whd, T* grad_P_x, T* grad_P_y,
                          T* grad_P_z, Tm1* du, LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    T dt_i = INFINITY;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) { break; }

        LocalIndex bodyBegin = grpStart[targetIdx];
        LocalIndex bodyEnd   = grpEnd[targetIdx];
        LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        if (i >= bodyEnd) continue;

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        T        maxvsignal;

        momentumAndEnergyJLoop<TravConfig::targetSize>(i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, vx, vy,
                                                       vz, h, m, rho, p, c, c11, c12, c13, c22, c23, c33, wh, whd,
                                                       grad_P_x, grad_P_y, grad_P_z, du, &maxvsignal);

        dt_i = stl::min(dt_i, tsKCourant(maxvsignal, h[i], c[i], Kcour));
    }

    typedef hipcub::BlockReduce<T, TravConfig::numThreads> BlockReduce;
    __shared__ typename BlockReduce::TempStorage        temp_storage;

    BlockReduce reduce(temp_storage);
    T           blockMin = reduce.Reduce(dt_i, hipcub::Min());
    __syncthreads();

    if (threadIdx.x == 0) { cstone::atomicMinFloat(&minDt_device, blockMin); }
}

template<class Dataset>
void computeMomentumEnergyStdGpu(const GroupView& grp, Dataset& d, const cstone::Box<typename Dataset::RealType>& box)
{
    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, d.ngmax);
    cstone::resetTraversalCounters<<<1, 1>>>();

    float huge = 1e10;
    checkGpuErrors(hipMemcpyToSymbol(GPU_SYMBOL(minDt_device), &huge, sizeof(huge)));
    cstone::resetTraversalCounters<<<1, 1>>>();

    cudaGradP<<<TravConfig::numBlocks(), TravConfig::numThreads>>>(
        d.K, d.Kcour, d.ngmax, box, grp.groupStart, grp.groupEnd, grp.numGroups, d.treeView, rawPtr(d.devData.x),
        rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.vx), rawPtr(d.devData.vy), rawPtr(d.devData.vz),
        rawPtr(d.devData.h), rawPtr(d.devData.m), rawPtr(d.devData.rho), rawPtr(d.devData.p), rawPtr(d.devData.c),
        rawPtr(d.devData.c11), rawPtr(d.devData.c12), rawPtr(d.devData.c13), rawPtr(d.devData.c22),
        rawPtr(d.devData.c23), rawPtr(d.devData.c33), rawPtr(d.devData.wh), rawPtr(d.devData.whd), rawPtr(d.devData.ax),
        rawPtr(d.devData.ay), rawPtr(d.devData.az), rawPtr(d.devData.du), nidxPool, traversalPool);

    checkGpuErrors(hipGetLastError());

    float minDt;
    checkGpuErrors(hipMemcpyFromSymbol(&minDt, HIP_SYMBOL(GPU_SYMBOL(minDt_device)), sizeof(minDt)));
    d.minDtCourant = minDt;
}

template void computeMomentumEnergyStdGpu(const GroupView& grp, sphexa::ParticlesData<cstone::GpuTag>& d,
                                          const cstone::Box<SphTypes::CoordinateType>&);
} // namespace sph
