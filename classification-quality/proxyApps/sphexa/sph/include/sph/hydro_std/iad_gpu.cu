#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Integral-approach-to-derivative i-loop GPU driver
 *
 * @author Ruben Cabezon <ruben.cabezon@unibas.ch>
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_std/iad_kern.hpp"

namespace sph
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

/*! @brief
 *
 * @tparam     T               float or double
 * @tparam     KeyType         32- or 64-bit unsigned integer
 * @param[in]  K
 * @param[in]  ngmax           maximum number of neighbors per particle to use
 * @param[in]  box             global coordinate bounding box
 * @param[in]  grpStart        start of each particle group, length @p numGroups
 * @param[in]  grpEnd          end of each particle groups, length @p numGroups
 * @param[in]  numGroups       number of groups
 * @param[in]  numParticles    number of local particles + halos
 * @param[in]  particleKeys    SFC keys of particles, sorted in ascending order
 * @param[in]  x               x coords, length @p numParticles, SFC sorted
 * @param[in]  y               y coords, length @p numParticles, SFC sorted
 * @param[in]  z               z coords, length @p numParticles, SFC sorted
 * @param[in]  h               smoothing lengths, length @p numParticles
 * @param[in]  m               masses, length @p numParticles
 * @param[in]  rho             densities, length @p numParticles
 * @param[in]  wh              sinc lookup table
 * @param[in]  whd             sinc derivative lookup table
 * @param[out] c11             output IAD components, length @p numParticles
 * @param[out] c12
 * @param[out] c13
 * @param[out] c22
 * @param[out] c23
 * @param[out] c33
 */
template<class Tc, class Tm, class T, class KeyType>
__global__ void IADGpuKernel(Tc K, unsigned ngmax, cstone::Box<Tc> box, const LocalIndex* grpStart,
                             const LocalIndex* grpEnd, LocalIndex numGroups,
                             const cstone::OctreeNsView<Tc, KeyType> tree, const Tc* x, const Tc* y, const Tc* z,
                             const T* h, const Tm* m, const T* rho, const T* wh, const T* whd, T* c11, T* c12, T* c13,
                             T* c22, T* c23, T* c33, LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) { return; }

        LocalIndex bodyBegin = grpStart[targetIdx];
        LocalIndex bodyEnd   = grpEnd[targetIdx];
        LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        if (i >= bodyEnd) { continue; }

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        sph::IADJLoopSTD<TravConfig::targetSize>(i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, h, m, rho, wh,
                                                 whd, c11, c12, c13, c22, c23, c33);
    }
}

template<class Dataset>
void computeIADGpu(const GroupView& grp, Dataset& d, const cstone::Box<typename Dataset::RealType>& box)
{
    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, d.ngmax);
    cstone::resetTraversalCounters<<<1, 1>>>();

    IADGpuKernel<<<TravConfig::numBlocks(), TravConfig::numThreads>>>(
        d.K, d.ngmax, box, grp.groupStart, grp.groupEnd, grp.numGroups, d.treeView, rawPtr(d.devData.x),
        rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.h), rawPtr(d.devData.m), rawPtr(d.devData.rho),
        rawPtr(d.devData.wh), rawPtr(d.devData.whd), rawPtr(d.devData.c11), rawPtr(d.devData.c12),
        rawPtr(d.devData.c13), rawPtr(d.devData.c22), rawPtr(d.devData.c23), rawPtr(d.devData.c33), nidxPool,
        traversalPool);
    checkGpuErrors(hipDeviceSynchronize());
}

template void computeIADGpu(const GroupView&, sphexa::ParticlesData<cstone::GpuTag>& d,
                            const cstone::Box<SphTypes::CoordinateType>&);

} // namespace sph
