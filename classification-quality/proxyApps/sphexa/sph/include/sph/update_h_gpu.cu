#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2022 CSCS, ETH Zurich
 *               2022 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Smoothing length update on the GPU
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/gpu_config.cuh"
#include "sph/sph_gpu.hpp"
#include "sph/kernels.hpp"

namespace sph
{
using cstone::LocalIndex;

template<class Th>
__global__ void updateSmoothingLengthGpuKernel(GroupView grp, unsigned ng0, const unsigned* nc, Th* h)
{
    LocalIndex laneIdx = threadIdx.x & (cstone::GpuConfig::warpSize - 1);
    LocalIndex warpIdx = (blockDim.x * blockIdx.x + threadIdx.x) >> cstone::GpuConfig::warpSizeLog2;
    if (warpIdx >= grp.numGroups) { return; }

    LocalIndex i = grp.groupStart[warpIdx] + laneIdx;
    if (i >= grp.groupEnd[warpIdx]) { return; }

    h[i] = updateH(ng0, nc[i], h[i]);
}

template<class Th>
void updateSmoothingLengthGpu(const GroupView& grp, unsigned ng0, const unsigned* nc, Th* h)
{
    unsigned numThreads       = 256;
    unsigned numWarpsPerBlock = numThreads / cstone::GpuConfig::warpSize;
    unsigned numBlocks        = (grp.numGroups + numWarpsPerBlock - 1) / numWarpsPerBlock;
    if (numBlocks == 0) { return; }
    updateSmoothingLengthGpuKernel<<<numBlocks, numThreads>>>(grp, ng0, nc, h);
}

template void updateSmoothingLengthGpu(const GroupView& grp, unsigned ng0, const unsigned* nc, float* h);
template void updateSmoothingLengthGpu(const GroupView& grp, unsigned ng0, const unsigned* nc, double* h);

} // namespace sph